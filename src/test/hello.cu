
#include <hip/hip_runtime.h>
#include "stdio.h"
#define N 100

__global__ void cuda_hello()
{
    printf("Hello World from GPU!\n");
}
__global__ void vector_add(float *out, float *a, float *b, int n)
{
    for (int i = 0; i < n; i++)
    {
        out[i] = a[i] + b[i];
    }
}
int main()
{
    printf("Hello World from CPU!\n");
    cuda_hello<<<1, 1>>>();
    // Allocate memory
    float *a, *b, *out;
    float *d_a, *d_b, *d_out;
    a = (float *)malloc(sizeof(float) * N);
    b = (float *)malloc(sizeof(float) * N);
    out = (float *)malloc(sizeof(float) * N);

    // Initialize array
    for (int i = 0; i < N; i++)
    {
        a[i] = 1.0f;
        b[i] = 2.0f;
    }
    // Allocate device memory for a
    hipMalloc((void **)&d_a, sizeof(float) * N);

    // Transfer data from host to device memory
    hipMemcpy(d_a, a, sizeof(float) * N, hipMemcpyHostToDevice);

    // Allocate device memory for b
    hipMalloc((void **)&d_b, sizeof(float) * N);

    // Transfer data from host to device memory
    hipMemcpy(d_b, b, sizeof(float) * N, hipMemcpyHostToDevice);

    // Allocate device memory for out
    hipMalloc((void **)&d_out, sizeof(float) * N);

    // Transfer data from host to device memory
    hipMemcpy(d_out, out, sizeof(float) * N, hipMemcpyHostToDevice);

    vector_add<<<1, 1>>>(d_out, d_a, d_b, N);
    hipMemcpy(out, d_out, sizeof(float) * N, hipMemcpyDeviceToHost);
    for (int i = 0; i < N; i++)
    {
        printf("%f \n", out[i]);
    }
    // Cleanup after kernel execution
    hipFree(d_a);
    free(a);
    hipFree(d_b);
    free(b);
    hipFree(d_out);
    free(out);

    return 0;
}